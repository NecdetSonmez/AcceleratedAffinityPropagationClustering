﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <string>
#include <chrono>

#include "ApcCpu.hpp"
#include "ApcGpu.hpp"
#include "ApcGpuV2.hpp"
#include "Points.hpp"

#define POINT_COUNT 1000
#define POINT_DIM 2

#define USE_CPU false
#define USE_GPU true
#define USE_GPU_V2 true

int main()
{
    // Generate points
    Points pointsObject;
    std::vector<std::tuple<float, float>> centers = {{0.0, 0.0}, {5.0, 5.0}};
    pointsObject.generatePoints(centers, POINT_COUNT, 0.25);
    float* points = pointsObject.getPoints();

#if USE_CPU
    // Cluster with CPU and measure time
    ApcCpu cpuClusterer(points, POINT_COUNT, POINT_DIM, 0.5);

    auto startTimeCpu = std::chrono::high_resolution_clock::now();
    cpuClusterer.cluster();
    auto endTimeCpu = std::chrono::high_resolution_clock::now();
    auto durationCpu = std::chrono::duration_cast<std::chrono::microseconds>(endTimeCpu - startTimeCpu).count();
    std::cout << "CPU Time taken: " << durationCpu << " microseconds." << std::endl;
#endif

#if USE_GPU
    // Cluster with GPU and measure time
    ApcGpu gpuClusterer(points, POINT_COUNT, POINT_DIM, 0.5);

    auto startTimeGpu = std::chrono::high_resolution_clock::now();
    gpuClusterer.cluster(25);
    auto endTimeGpu = std::chrono::high_resolution_clock::now();
    auto durationGpu = std::chrono::duration_cast<std::chrono::microseconds>(endTimeGpu - startTimeGpu).count();
    std::cout << "GPU Time taken: " << durationGpu << " microseconds." << std::endl;
#endif

#if USE_GPU_V2
    // Cluster with GPU and measure time
    ApcGpuV2 gpuClustererV2(points, POINT_COUNT, POINT_DIM, 0.5);

    auto startTimeGpuV2 = std::chrono::high_resolution_clock::now();
    gpuClustererV2.cluster(25);
    auto endTimeGpuV2 = std::chrono::high_resolution_clock::now();
    auto durationGpuV2 = std::chrono::duration_cast<std::chrono::microseconds>(endTimeGpuV2 - startTimeGpuV2).count();
    std::cout << "GPU Time taken: " << durationGpuV2 << " microseconds." << std::endl;
#endif

    return 0;
}