﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <string>
#include <chrono>

#include "ApcCpu.hpp"
#include "ApcGpu.hpp"
#include "ApcGpuV2.hpp"
#include "Points.hpp"

#define POINT_COUNT 100
#define POINT_DIM 2
#define POINT_VARIATION 0.25

#define DAMPING_FACTOR 0.5
#define ITERATION_COUNT 100

#define USE_CPU true
#define USE_GPU true
#define USE_GPU_V2 true

int main()
{
    // Generate points
    Points pointsObject;
    std::vector<std::tuple<float, float>> centers = {{0.0, 0.0}, {5.0, 5.0}};
    //std::vector<std::tuple<float, float>> centers = {{0.0, 0.0}};
    pointsObject.generatePoints(centers, POINT_COUNT, POINT_VARIATION);
    float* points = pointsObject.getPoints();

#if USE_CPU
    // Cluster with CPU and measure time
    ApcCpu cpuClusterer(points, POINT_COUNT, POINT_DIM, DAMPING_FACTOR);

    auto startTimeCpu = std::chrono::high_resolution_clock::now();
    cpuClusterer.cluster(ITERATION_COUNT);
    auto endTimeCpu = std::chrono::high_resolution_clock::now();
    auto durationCpu = std::chrono::duration_cast<std::chrono::microseconds>(endTimeCpu - startTimeCpu).count();
    std::cout << "CPU Time taken: " << durationCpu << " microseconds." << std::endl;
#endif

#if USE_GPU
    // Cluster with GPU and measure time
    ApcGpu gpuClusterer(points, POINT_COUNT, POINT_DIM, DAMPING_FACTOR);

    auto startTimeGpu = std::chrono::high_resolution_clock::now();
    gpuClusterer.cluster(ITERATION_COUNT);
    auto endTimeGpu = std::chrono::high_resolution_clock::now();
    auto durationGpu = std::chrono::duration_cast<std::chrono::microseconds>(endTimeGpu - startTimeGpu).count();
    std::cout << "GPU Time taken: " << durationGpu << " microseconds." << std::endl;
#endif

#if USE_GPU_V2
    // Cluster with GPU and measure time
    ApcGpuV2 gpuClustererV2(points, POINT_COUNT, POINT_DIM, DAMPING_FACTOR);

    auto startTimeGpuV2 = std::chrono::high_resolution_clock::now();
    gpuClustererV2.cluster(ITERATION_COUNT);
    auto endTimeGpuV2 = std::chrono::high_resolution_clock::now();
    auto durationGpuV2 = std::chrono::duration_cast<std::chrono::microseconds>(endTimeGpuV2 - startTimeGpuV2).count();
    std::cout << "GPU Time taken: " << durationGpuV2 << " microseconds." << std::endl;
#endif

    return 0;
}