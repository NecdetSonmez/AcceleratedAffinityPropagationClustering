﻿
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#include "AffinityPropagationClustering.hpp"

int main()
{
    // Test points for the algorithm
    float points[] = {1.1, 2.0, 1.0, 2.1, 0.9, 2.0, 1.2, 2.1, 3.0, 4.5, 3.1, 4.3, 3.2, 4.3, 3.4, 4.4};
    AffinityPropagationClustering a(points, 8, 2, 0.5);
    a.clusterCpu();
    return 0;
}